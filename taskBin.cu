#include "taskBin.h"
#include <iostream>

// Default constructor
TaskBin::TaskBin() : M_pad(0), N_pad(0), R_pad(0) {
    hipStreamCreate(&stream);
}

// Constructor to initialize bin dimensions and CUDA stream
TaskBin::TaskBin(int M_pad, int N_pad, int R_pad)
    : M_pad(M_pad), N_pad(N_pad), R_pad(R_pad) {
    hipStreamCreate(&stream);
}

// Destructor to destroy CUDA stream
TaskBin::~TaskBin() {
    hipStreamDestroy(stream);
}

// Method to add a task to the bin
void TaskBin::addTask(const Task &task) {
    tasks.push_back(task);
}

// Method to pad and compute matrix multiplication using cuBLAS batch mode
void TaskBin::padAndCompute(hipblasHandle_t &handle) {
    int batchSize = tasks.size();
    std::vector<const hipComplex *> d_A(batchSize);
    std::vector<const hipComplex *> d_B(batchSize);
    std::vector<hipComplex *> d_C(batchSize);

    for (int i = 0; i < batchSize; ++i) {
        d_A[i] = tasks[i].d_Qmat;
        d_B[i] = tasks[i].d_Rmat;
        d_C[i] = tasks[i].d_localMat;
    }

    const hipComplex alpha = make_hipComplex(1.0f, 0.0f);
    const hipComplex beta = make_hipComplex(0.0f, 0.0f);

    hipblasSetStream(handle, stream);
    hipblasCgemmBatched(handle,
                       HIPBLAS_OP_N, HIPBLAS_OP_N,
                       M_pad, R_pad, N_pad,
                       &alpha,
                       d_A.data(), M_pad,
                       d_B.data(), N_pad,
                       &beta,
                       d_C.data(), M_pad,
                       batchSize);
}
