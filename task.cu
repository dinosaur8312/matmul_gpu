#include "task.h"
#include <hip/hip_runtime.h>
#include <iostream>

// Constructor to initialize task dimensions and compute padded dimensions
Task::Task(int M, int N, int R)
    : M(M), N(N), R(R),
      M_pad((M + 15) / 16 * 16), N_pad((N + 15) / 16 * 16), R_pad((R + 15) / 16 * 16),
      d_pDense(nullptr), d_Qmat(nullptr), d_Rmat(nullptr),
      d_localB(nullptr), d_localC(nullptr), d_localMat(nullptr) {}

// Destructor to free GPU memory
Task::~Task() {
    hipFree(d_pDense);
    hipFree(d_Qmat);
    hipFree(d_Rmat);
    hipFree(d_localB);
    hipFree(d_localC);
    hipFree(d_localMat);
}

// Method to allocate GPU memory for task matrices using padded dimensions
void Task::allocateDeviceMemory(int nRHS) {
    hipMalloc(&d_pDense, M_pad * N_pad * sizeof(hipComplex));
    hipMalloc(&d_Qmat, M_pad * R_pad * sizeof(hipComplex));
    hipMalloc(&d_Rmat, R_pad * N_pad * sizeof(hipComplex));
    hipMalloc(&d_localB, N_pad * nRHS * sizeof(hipComplex));
    hipMalloc(&d_localC, M_pad * nRHS * sizeof(hipComplex));
    hipMalloc(&d_localMat, M_pad * N_pad * sizeof(hipComplex));
}
