#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <tuple>
#include <map>

#include "kernel.cuh"

#define CUDA_CHECK(call)                                                                                           \
    do                                                                                                             \
    {                                                                                                              \
        hipError_t err = call;                                                                                    \
        if (err != hipSuccess)                                                                                    \
        {                                                                                                          \
            fprintf(stderr, "CUDA error in %s (%s:%d): %s\n", #call, __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE);                                                                                    \
        }                                                                                                          \
    } while (0)

#define CUBLAS_CHECK(call)                                                              \
    do                                                                                  \
    {                                                                                   \
        hipblasStatus_t status = call;                                                   \
        if (status != HIPBLAS_STATUS_SUCCESS)                                            \
        {                                                                               \
            fprintf(stderr, "cuBLAS error in %s (%s:%d)\n", #call, __FILE__, __LINE__); \
            exit(EXIT_FAILURE);                                                         \
        }                                                                               \
    } while (0)

const int group_count = 3;
const int batch_sizes[group_count] = {8192};
const int m[group_count] = {32, 64, 128};
const int n[group_count] = {32, 64, 128};
const int k[group_count] = {32, 64, 128};

const int matrix_dim_global = 1024;

using data_type = hipComplex;

void initialize_matrix(data_type **matrices, int rows, int cols, int batch_num)
{
    data_type value = make_hipComplex(1.f, 0.f);
    for (int ibatch = 0; ibatch < batch_num; ibatch++)
    {
        for (int i = 0; i < rows * cols; i++)
        {
            matrices[ibatch][i] = value;
        }
    }
}

void print_matrix(int rows, int cols, const data_type *matrix, int ld)
{
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            hipComplex val = matrix[i + j * ld];
            printf("(%f, %f) ", hipCrealf(val), hipCimagf(val));
        }
        printf("\n");
    }
}

std::vector<std::tuple<int, int, int>> parseMILFile(const std::string &filePath)
{
    std::vector<std::tuple<int, int, int>> parsedData;
    std::ifstream file(filePath);
    if (!file.is_open())
    {
        std::cerr << "Error: Unable to open file " << filePath << std::endl;
        return parsedData;
    }

    std::string line;
    while (std::getline(file, line))
    {
        std::istringstream lineStream(line);
        std::string word;
        std::vector<std::string> words;
        while (lineStream >> word)
        {
            words.push_back(word);
        }

        // Check if the line has at least 7 elements (based on the given format)
        if (words.size() >= 7)
        {
            int M = std::stoi(words[4]);
            int N = std::stoi(words[5]);
            int R = std::stoi(words[6]);
            parsedData.emplace_back(M, N, R);
        }
    }

    file.close();
    return parsedData;
}

std::map<std::tuple<int, int, int>, std::vector<std::tuple<int, int, int>>>
binByMultipleOf16(const std::vector<std::tuple<int, int, int>> &data)
{
    std::map<std::tuple<int, int, int>, std::vector<std::tuple<int, int, int>>> bins;

    for (const auto &entry : data)
    {
        int M = std::get<0>(entry);
        int N = std::get<1>(entry);
        int R = std::get<2>(entry);

        // Calculate the bin key
        int binM = (M+15) / 16;
        int binN = (N+15) / 16;
        int binR = (R+15) / 16;
        std::tuple<int, int, int> binKey = std::make_tuple(binM, binN, binR);

        // Insert the entry into the appropriate bin
        bins[binKey].emplace_back(M, N, R);
    }

    return bins;
}

// Helper function to print the bins
void printBins(const std::map<std::tuple<int, int, int>, std::vector<std::tuple<int, int, int>>> &bins)
{
    for (const auto &[key, values] : bins)
    {
        std::cout << "Bin (" << std::get<0>(key) << ", " << std::get<1>(key) << ", " << std::get<2>(key) << "):\n";
        for (const auto &value : values)
        {
            std::cout << "    M: " << std::get<0>(value)
                      << ", N: " << std::get<1>(value)
                      << ", R: " << std::get<2>(value) << "\n";
        }
    }
}

// Helper function to print the size of each bin
void printBinsSize(const std::map<std::tuple<int, int, int>, std::vector<std::tuple<int, int, int>>> &bins) {
    std::cout << "Bin Sizes:\n";
    for (const auto &[key, values] : bins) {
        std::cout << "Bin (" << std::get<0>(key)*16 << ", " << std::get<1>(key)*16 << ", " << std::get<2>(key)*16 << "): "
                  << values.size() << " elements\n";
    }
}

// Function to split the bins into two maps: R = 0 and R > 0
std::pair<
    std::map<std::tuple<int, int, int>, std::vector<std::tuple<int, int, int>>>,
    std::map<std::tuple<int, int, int>, std::vector<std::tuple<int, int, int>>>>
splitBins(const std::map<std::tuple<int, int, int>, std::vector<std::tuple<int, int, int>>> &bins) {
    std::map<std::tuple<int, int, int>, std::vector<std::tuple<int, int, int>>> binsR0;
    std::map<std::tuple<int, int, int>, std::vector<std::tuple<int, int, int>>> binsRPositive;

    for (const auto &[key, values] : bins) {
        for (const auto &value : values) {
            int R = std::get<2>(value);
            if (R == 0) {
                binsR0[key].emplace_back(value);
            } else {
                binsRPositive[key].emplace_back(value);
            }
        }
    }

    return {binsR0, binsRPositive};
}


int main()
{

    const std::string filePath = "MIL_Thread_combo_small.txt";
    auto data = parseMILFile(filePath);
     auto bins = binByMultipleOf16(data);
    //printBins(bins);
   // printBinsSize(bins);

    auto [binsR0, binsRPositive] = splitBins(bins);

    std::cout << "Bins with R = 0:\n";
    printBinsSize(binsR0);

    std::cout << "\nBins with R > 0:\n";
    printBinsSize(binsRPositive);


    hipblasHandle_t cublasH[group_count];
    hipStream_t streams[group_count];

    data_type **A[group_count], **B[group_count], **C[group_count];
    data_type **d_A[group_count], **d_B[group_count], **d_C[group_count];
    data_type **d_A_array[group_count], **d_B_array[group_count], **d_C_array[group_count];

    data_type *d_A_global, *d_B_global, *d_C_global;

    CUDA_CHECK(hipMalloc((void **)&d_A_global, sizeof(data_type) * matrix_dim_global * matrix_dim_global));
    CUDA_CHECK(hipMalloc((void **)&d_B_global, sizeof(data_type) * matrix_dim_global * matrix_dim_global));
    CUDA_CHECK(hipMalloc((void **)&d_C_global, sizeof(data_type) * matrix_dim_global * matrix_dim_global));

    for (int g = 0; g < group_count; g++)
    {
        int batch_size = batch_sizes[g];
        int lm = m[g], ln = n[g], lk = k[g];
        A[g] = new data_type *[batch_size];
        B[g] = new data_type *[batch_size];
        C[g] = new data_type *[batch_size];

        for (int i = 0; i < batch_size; i++)
        {
            A[g][i] = new data_type[lm * lk];
            B[g][i] = new data_type[lk * ln];
            C[g][i] = new data_type[lm * ln];
        }

        initialize_matrix(A[g], lm, lk, batch_size);
        initialize_matrix(B[g], lk, ln, batch_size);
    }

    // print matrix
    /*
    for (int g = 0; g < group_count; g++)
    {
        int batch_num =  batch_sizes[g];
        for (int i = 0; i < batch_num; i++)
        {
            printf("Group %d, Matrix %d:\n", g, i);
            print_matrix(m[g], k[g], A[g][i], m[g]);
            print_matrix(k[g], n[g], B[g][i], k[g]);
        }
    }
*/
    const data_type alpha = make_hipComplex(1.f, 0.f);
    const data_type beta = make_hipComplex(0.f, 0.f);

    for (int g = 0; g < group_count; g++)
    {
        CUDA_CHECK(hipStreamCreate(&streams[g]));
        CUBLAS_CHECK(hipblasCreate(&cublasH[g]));
        CUBLAS_CHECK(hipblasSetStream(cublasH[g], streams[g]));

        int batch_num = batch_sizes[g];

        // d_A, d_B, d_C are device pointers of size batch_num to store the device pointers of batch_num matrices

        d_A[g] = new data_type *[batch_num];
        d_B[g] = new data_type *[batch_num];
        d_C[g] = new data_type *[batch_num];

        CUDA_CHECK(hipMalloc((void **)&d_A_array[g], sizeof(data_type *) * batch_num));
        CUDA_CHECK(hipMalloc((void **)&d_B_array[g], sizeof(data_type *) * batch_num));
        CUDA_CHECK(hipMalloc((void **)&d_C_array[g], sizeof(data_type *) * batch_num));

        for (int i = 0; i < batch_num; i++)
        {
            size_t A_size = sizeof(data_type) * m[g] * k[g];
            size_t B_size = sizeof(data_type) * k[g] * n[g];
            size_t C_size = sizeof(data_type) * m[g] * n[g];

            CUDA_CHECK(hipMalloc((void **)&d_A[g][i], sizeof(data_type) * A_size));
            CUDA_CHECK(hipMalloc((void **)&d_B[g][i], sizeof(data_type) * B_size));
            CUDA_CHECK(hipMalloc((void **)&d_C[g][i], sizeof(data_type) * C_size));

            CUDA_CHECK(hipMemcpyAsync(d_A[g][i], A[g][i], A_size, hipMemcpyHostToDevice, streams[g]));
            CUDA_CHECK(hipMemcpyAsync(d_B[g][i], B[g][i], B_size, hipMemcpyHostToDevice, streams[g]));

            // d_Ai, d_Bi, d_Ci are device pointers, but in host memory.
            // CUDA_CHECK(hipMemcpyAsync(&d_A[g][i], &d_Ai, sizeof(data_type *), hipMemcpyHostToDevice, streams[g]));
            // CUDA_CHECK(hipMemcpyAsync(&d_B[g][i], &d_Bi, sizeof(data_type *), hipMemcpyHostToDevice, streams[g]));
            // CUDA_CHECK(hipMemcpyAsync(&d_C[g][i], &d_Ci, sizeof(data_type *), hipMemcpyHostToDevice, streams[g]));

            // printf("Group %d, Matrix %d, d_Ci=%p, d_C[g]=%p\n", g, i, d_Ci, d_C[g]);

            // print d_A, A_B, d_C
            // printf("Group %d, Matrix %d, d_A=%p, d_B=%p, d_C=%p\n", g, i, d_A[g][i], d_B[g][i], d_C[g][i]);
        }

        // copy d_A, d_B, d_C to device memory
        CUDA_CHECK(hipMemcpyAsync(d_A_array[g], d_A[g], sizeof(data_type *) * batch_num, hipMemcpyHostToDevice, streams[g]));
        CUDA_CHECK(hipMemcpyAsync(d_B_array[g], d_B[g], sizeof(data_type *) * batch_num, hipMemcpyHostToDevice, streams[g]));
        CUDA_CHECK(hipMemcpyAsync(d_C_array[g], d_C[g], sizeof(data_type *) * batch_num, hipMemcpyHostToDevice, streams[g]));

        // d_A, d_B, d_C are device pointers, but their inside values, such as d_C[g][i] are in device memory, which cannot be access directly in host code
    }

    for (int g = 0; g < group_count; g++)
    {
        CUDA_CHECK(hipStreamSynchronize(streams[g]));
    }

    hipDeviceSynchronize();
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    CUDA_CHECK(hipEventRecord(start));

    for (int g = 0; g < group_count; g++)
    {

        dim3 threads_scatter = dim3(32, 32);
        dim3 blocks_scatter = dim3((m[g] + threads_scatter.x - 1) / threads_scatter.x, (k[g] + threads_scatter.y - 1) / threads_scatter.y, batch_sizes[g]);

        matrix_scatter<<<blocks_scatter, threads_scatter, 0, streams[g]>>>(d_A_global, d_A_array[g], m[g], k[g], matrix_dim_global);

        CUBLAS_CHECK(hipblasCgemmBatched(
            cublasH[g],
            HIPBLAS_OP_N, HIPBLAS_OP_N,
            m[g], n[g], k[g],
            &alpha,
            d_A_array[g], m[g],
            d_B_array[g], k[g],
            &beta,
            d_C_array[g], m[g],
            batch_sizes[g]));

        dim3 threads_gather = dim3(32, 32);
        dim3 blocks_gather = dim3((m[g] + threads_gather.x - 1) / threads_gather.x, (n[g] + threads_gather.y - 1) / threads_gather.y, batch_sizes[g]);

        matrix_gather<<<blocks_gather, threads_gather, 0, streams[g]>>>(d_C_global, d_C_array[g], m[g], n[g], matrix_dim_global);
    }

    hipDeviceSynchronize();

    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    float milliseconds = 0;
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    printf("Time: %f ms\n", milliseconds);
    // calculate GFLOPS
    size_t total_flops = 0;
    for (int g = 0; g < group_count; g++)
    {
        total_flops += 6 * m[g] * n[g] * k[g] * batch_sizes[g];
    }
    printf("Total FLOPS: %lu\n", total_flops);
    float gflop = total_flops / 1e9;
    printf("GFLOP: %f\n", total_flops / 1e9);

    float gflops = gflop * 1e3 / (milliseconds);
    printf("GFLOP/S: %f\n", gflops);

    /*


    for (int g = 0; g < group_count; g++)
    {
        int batch_num =  batch_sizes[g];
        for (int i = 0; i < batch_num; i++)
        {
            printf("Group %d, Matrix %d:\n", g, i);
            printf("C[%d][%d]=%p\n", g, i, C[g][i]);
            printf("d_C[%d][%d]=%p\n", g, i, d_C[g][i]);
            CUDA_CHECK(hipMemcpyAsync(C[g][i], d_C[g][i], sizeof(data_type) * m[g] * n[g], hipMemcpyDeviceToHost, streams[g]));
        }
    }

    for (int g = 0; g < group_count; g++)
    {
        CUDA_CHECK(hipStreamSynchronize(streams[g]));

        for (int i = 0; i < group_sizes[g]; i++)
        {
            printf("Group %d, Matrix %d:\n", g, i);
            print_matrix(m[g], n[g], C[g][i], m[g]);
        }

        for (int i = 0; i < group_sizes[g]; i++)
        {
            delete[] A[g][i];
            delete[] B[g][i];
            delete[] C[g][i];
        }

        delete[] A[g];
        delete[] B[g];
        delete[] C[g];

        hipFree(d_A[g]);
        hipFree(d_B[g]);
        hipFree(d_C[g]);

        hipblasDestroy(cublasH[g]);
        hipStreamDestroy(streams[g]);
    }
    */
    return EXIT_SUCCESS;
}
